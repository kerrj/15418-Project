#include "hip/hip_runtime.h"
#include "harris.h"

#include <stdio.h>
#include <chrono>
#include <string>
#include <hip/hip_fp16.h>
const int window = 9;
const int halfwin=window/2;

/* Helper function to round up to a power of 2. 
 */
static inline int nextPow2(int n)
{
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}
const int BDIM = 32;
template<typename T>
inline __device__ T& block2Buf(T* blockBuf, const int x, const int y, const int pad_size){
	return blockBuf[pad_size + x + (pad_size + y)*(BDIM+pad_size*2)];
}
inline __device__ float getVal(const half* blockBuf,const float* im,const int im_w, const int x,const int y,const int pad_size){
	if(x<0 || y<0 || x>=BDIM || y>=BDIM){
		return im[(blockDim.y*blockIdx.y+y)*im_w + blockDim.x*blockIdx.x+x];
	}else{
		return __half2float(blockBuf[pad_size + x + (pad_size + y)*(BDIM+pad_size*2)]);
	}
}
inline __device__ void fillSquare(half* blockBuf, const float* im, const int x, const int y, const int pad_size, const int im_w, const int im_h,
		const int globalX,const int globalY){
	for(int dy = 0; dy < pad_size; dy++) {
		if(globalY + dy >= im_h || globalY + dy < 0) continue;
		for(int dx = 0; dx < pad_size; dx++) {
			if(globalX + dx >= im_w || globalX + dx < 0) continue;
			block2Buf(blockBuf, x + dx, y + dy, pad_size) = __float2half(im[globalX + dx + (globalY + dy)*im_w]);
		}
	}
}

inline __device__ void fillBuf(half* blockBuf,const float* im, const int im_w, const int im_h, const int pad_size, int globalX, int globalY){
	//pad size is the size of the border we need to load in (should be window/2 for kernels)
	//we guarantee that globalX and globalY will be in bounds
	//first fill the middle
	block2Buf(blockBuf,threadIdx.x,threadIdx.y,pad_size) = __float2half(im[globalX + im_w*globalY]);
	//then handle the edges
	
	bool corner=false;
	int tlx,tly;
	if(threadIdx.x == 0){
		//left col
		if(threadIdx.y==0){
			//top left
			corner=true;
			tlx=-pad_size;
			tly=-pad_size;
		}
		for(int i = -1;i >= -pad_size;i--){
			if(globalX+i<0)break;
			block2Buf(blockBuf,threadIdx.x + i,threadIdx.y,pad_size) = __float2half(im[globalX + i + globalY*im_w]);
		}
	}else if(threadIdx.y == 0){
		//top row
		if(threadIdx.x==blockDim.x-1){
			//top right
			corner=true;
			tlx=blockDim.x;
			tly=-pad_size;
		}
		for(int i = -1;i >= -pad_size;i--){
			if(globalY+i<0)break;
			block2Buf(blockBuf,threadIdx.x,threadIdx.y+i,pad_size) = __float2half(im[globalX + (globalY + i)*im_w]);
		}
	}else if(threadIdx.x == blockDim.x-1){
		//right col
		if(threadIdx.y==blockDim.y-1){
			//bot right
			corner=true;
			tlx=blockDim.x;
			tly=blockDim.y;
		}
		for(int i = 1;i <= pad_size;i++){
			if(globalX+i>=im_w)break;
			block2Buf(blockBuf,threadIdx.x + i,threadIdx.y,pad_size) = __float2half(im[globalX + i + globalY*im_w]);
		}
	}else if(threadIdx.y == blockDim.y-1){
		//bot row
		if(threadIdx.x==0){
			//bot left
			corner=true;
			tlx=-pad_size;
			tly=blockDim.y;
		}
		for(int i = 1;i <= pad_size;i++){
			if(globalY+i>=im_h)break;
			block2Buf(blockBuf,threadIdx.x,threadIdx.y+i,pad_size) = __float2half(im[globalX + (globalY + i)*im_w]);
		}
	}
	if(corner){
		fillSquare(blockBuf,im,tlx,tly,pad_size,im_w,im_h,globalX,globalY);
	}
	
}
const int BUFSIZE=(BDIM + 2*halfwin)*(BDIM + 2*halfwin);
__global__ void _harrisActivation(const float* __restrict__ gradX, const float*  __restrict__ gradY,const int img_w,const int img_h, float* __restrict__ output) {
	__shared__ half gradXBuf[BUFSIZE];
	__shared__ half gradYBuf[BUFSIZE];
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	if(x - halfwin < 0 || x + halfwin >= img_w 
		|| y - halfwin < 0 || y + halfwin >= img_h){
		 __syncthreads();
		 return;
	}
	fillBuf(gradXBuf,gradX,img_w,img_h,halfwin,x,y);
	fillBuf(gradYBuf,gradY,img_w,img_h,halfwin,x,y);
	__syncthreads();
	
	half valx=0,valy=0,valz=0; // Ixx, Iyy, Ixy
	const int tilew=(BDIM+halfwin*2);
	for(int r = 0; r < window; r++) {
		//const int newy = y + r - halfwin;
		const int newy = threadIdx.y+r-halfwin;
		for(int c = 0; c < window; c++) {
			//int newx = x + c - halfwin;
			//const float gX = gradX[newy*img_w + newx];
			//const float gY = gradY[newy*img_w + newx];
			
			const int newx = threadIdx.x+c-halfwin;
			const int bufid = halfwin + newx + (halfwin + newy)*tilew;
			const half gX = (gradXBuf[bufid]);
			const half gY = (gradYBuf[bufid]);
			
			//const float gX = getVal(gradXBuf,gradX,img_w,newx,newy,halfwin);
			//const float gY = getVal(gradYBuf,gradY,img_w,newx,newy,halfwin);
			valx += gX * gX;
			valy += gY * gY;
			valz += gX * gY;
		}
	}
	// Compute R = det(M) - k * tr(M)^2
	const float det = __half2float(valx * valy - valz * valz);
	const float R=det - 0.05 * __half2float((valx + valy) * (valx + valy));
	const int keepflag = R>0.3;
	output[img_w * y + x] = R*keepflag;
}

const int nmswindow=3;
__global__ void _nms(const float* __restrict__ activations, int img_w, int img_h, unsigned short* __restrict__ threshold_output) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	
	threshold_output[y*img_w + x] = 0;
	if(x - (window/2) < 0 || x + (window/2) >= img_w) return;
	if(y - (window/2) < 0 || y + (window/2) >= img_h) return;
	
	const float val = activations[y*img_w + x];
	if(val < 0.001) return;
	for(int r = 0; r < nmswindow; r++) {
		int newy = y + r - (nmswindow / 2);
		for(int c = 0; c < nmswindow; c++) {
			int newx = x + c - (nmswindow / 2);
			const float newVal = activations[newy*img_w + newx];
			// Returns since value is 0 if there exists a higher-activation neighbor
			if(newVal > val) {
				return;
			}
		}
	}
	threshold_output[y*img_w + x] = 1;
}

__global__
void scan_down_kernel(unsigned short* device_data, int size, int device_data_size_of_array_yes_sir_not_the_previous_size) {
    const int index = (blockIdx.x * blockDim.x + threadIdx.x+1) * size - 1;
    // Check if index is going to be written to at this level
    if(index < device_data_size_of_array_yes_sir_not_the_previous_size) {
        // Get the index it will be combined with (halfway) 
        const int j = index - (size >> 1);
        device_data[index] = device_data[index] + device_data[j];
    }
}

__global__
void scan_up_kernel(unsigned short* device_data, int size, int device_data_size_of_array_yes_sir_not_the_previous_size) {
    //int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int index = (blockIdx.x * blockDim.x + threadIdx.x+1) * size - 1;
    // if(index % size == size - 1) {
    if(index<device_data_size_of_array_yes_sir_not_the_previous_size){
        const int j = index - (size >> 1);
        const short original = device_data[index];
        device_data[index] += device_data[j];
        device_data[j] = original;
    }
}


__global__ void _collapse(const unsigned short* __restrict__ scanResult, const float* __restrict__ activations, int size, unsigned int* __restrict__ locations, float* __restrict__ outputActivations) {
	const int index = blockIdx.x*blockDim.x + threadIdx.x + 1;
    if(index>=size)return;
    if(scanResult[index]>scanResult[index-1]){
        //this is a new index to count in output
        locations[scanResult[index-1]] = index-1;
        outputActivations[scanResult[index-1]] = activations[scanResult[index-1]];
    }
}
// Timing 
std::chrono::time_point<std::chrono::system_clock> tic2(){
	return std::chrono::system_clock::now();
}
double toc2(std::string msg,std::chrono::time_point<std::chrono::system_clock> & t){
	std::chrono::duration<double> elapsed=std::chrono::system_clock::now()-t;
	printf("Time for %s: %f\n",msg.c_str(),elapsed.count());
	t = tic2();
	return elapsed.count();
}
void harris(float* gradX, float* gradY, int img_w, int img_h, float* activations, unsigned short* threshold_output) {
	// Requires that img and output are hipMalloc'd by caller
	const dim3 blockSize(BDIM, BDIM);
	// Make Gridsize
	const dim3 gridDims((img_w + blockSize.x - 1) / blockSize.x,
                 (img_h + blockSize.y - 1) / blockSize.y);
    auto t=tic2();
	_harrisActivation<<<gridDims, blockSize>>>(gradX, gradY, img_w, img_h, activations);
	hipDeviceSynchronize();
	toc2("harris kern",t);
	_nms<<<gridDims, blockSize>>>(activations, img_w, img_h, threshold_output);
}

void scan(unsigned short* device_data, int length)
{
    const int threadsPerBlock = 64;
    const int length_nextPow2 = nextPow2(length);
    int i = 2;
    for (; i < length_nextPow2; i=(i<<1)) {
        const size_t n = length_nextPow2/i;//num elements to compute
        const int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;
        scan_down_kernel<<<blocks, threadsPerBlock>>>(device_data, i, length_nextPow2);
    }
    const short x = 0;
    hipMemcpy(&device_data[length_nextPow2-1], &x, sizeof(short), hipMemcpyHostToDevice);
    for(; i > 1; i=(i>>1)) {
        const size_t n = length_nextPow2/i;//num elements to compute
        const int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;
        //const int blocks = (length_nextPow2 + threadsPerBlock - 1) / threadsPerBlock;
        scan_up_kernel<<<blocks, threadsPerBlock>>>(device_data, i,length_nextPow2);
    }
}


void collapse(const unsigned short* scanResult, const float* activations, int size, unsigned int* locations, float* outputActivations) {
	const int threadsPerBlock = 64;
	const int blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
	_collapse<<<blocks, threadsPerBlock>>>(scanResult, activations, size, locations, outputActivations);	
}

bool compareCorners(std::pair<float, unsigned int> c1, std::pair<float, unsigned int> c2) {
	return c1.first > c2.first;
}

std::vector<unsigned int> selectCorners(unsigned int* locations, float* outputActivations, int numCorners, int numSelect) {
	// numCorners: Number of corners passed in
	// numSelect: Number of corners to return 
	// Sorts locations by outputActivations (outputActivations is unchanged)
	// Copy data from cuda memory
	std::vector<unsigned int> locations_local(numCorners);
	std::vector<float> activations_local(numCorners);
	hipMemcpy(locations_local.data(), locations, sizeof(int)*numCorners, hipMemcpyDeviceToHost);
	hipMemcpy(activations_local.data(), outputActivations, sizeof(float)*numCorners, hipMemcpyDeviceToHost);
	// Make pairs of (activation, location)
	std::vector<std::pair<float, unsigned int> > cornerPairs(numCorners);
	for(int i = 0; i < numCorners; i++)
		cornerPairs[i] = std::make_pair(activations_local[i], locations_local[i]);  
	// Do partial sort 
	std::partial_sort(cornerPairs.begin(), cornerPairs.begin() + numSelect, cornerPairs.end(), compareCorners);
	// Return locations only as a vector 
	for(int i = 0; i < numSelect; i++)
		locations_local[i] = cornerPairs[i].second;
	hipMemcpy(locations, locations_local.data(), sizeof(int)*numSelect, hipMemcpyHostToDevice);
	return locations_local;
};

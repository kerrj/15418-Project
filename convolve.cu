#include "hip/hip_runtime.h"
#include "convolve.h"

__global__ void _convolve(float* img, int img_w, int img_h, float* kernel, int kernel_w, int kernel_h, float* output) {
}

template<int kernel_w, int kernel_h>
 void Convolve<kernel_w, kernel_h>::doConvolve(float* img, int img_w, int img_h, float* output) {
	// Requires that img and output are hipMalloc'd by caller
	dim3 blockSize(32, 32);
	// Make Gridsize
	dim3 gridDims((img_w + blockSize.x - 1) / blockSize.x,
                 (img_h + blockSize.y - 1) / blockSize.y);
                 
	_convolve<<<gridDims, blockSize>>>(img, img_w, img_h, kernel, kernel_w, kernel_h, output);
}

//um wait holdon
//forward declare the templates we will need later
template class Convolve<3,3>;
